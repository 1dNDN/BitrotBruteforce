#include "hip/hip_runtime.h"
#include "sha1.cuh"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif

constexpr auto K0 = 0x5a827999;
constexpr auto K1 = 0x6ed9eba1;
constexpr auto K2 = 0x8f1bbcdc;
constexpr auto K3 = 0xca62c1d6;

__host__ __device__ void sha1_transform(SHA1_CTX* ctx, const BYTE data[])
{
	WORD a, b, c, d, e, i, j, t, m[80];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) + (data[j + 1] << 16) + (data[j + 2] << 8) + (data[j + 3]);

	for (; i < 80; ++i) {
		m[i] = (m[i - 3] ^ m[i - 8] ^ m[i - 14] ^ m[i - 16]);
		m[i] = (m[i] << 1) | (m[i] >> 31);
	}

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];

	for (i = 0; i < 20; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (~b & d)) + e + K0 + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	for (; i < 40; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + K1 + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	for (; i < 60; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (b & d) ^ (c & d)) + e + K2 + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	for (; i < 80; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + K3 + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
}

__host__ __device__ void sha1_init(SHA1_CTX* ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xc3d2e1f0;
}

__host__ __device__ void sha1_update(SHA1_CTX* ctx, const BYTE data[], size_t len)
{
	size_t i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			sha1_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__host__ __device__ void sha1_final(SHA1_CTX* ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	ctx->data[0] = 0x80;
	memset(&ctx->data[1], 0x00, 55);

	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	sha1_transform(ctx, ctx->data);

	for (i = 0; i < 4; ++i) {
		hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
	}
}

__global__ void bitFlipKernel(unsigned char* pieceData, unsigned char* pieceHash, SHA1_CTX* midstates, size_t fileSize, unsigned int* result)
{
	unsigned int batchIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (batchIdx < fileSize / BATCH_SIZE && *result == -1) {
		unsigned int startingBit = batchIdx * BATCH_SIZE * 8;
		unsigned int endingBit;

		if (startingBit + BATCH_SIZE * 8 < fileSize * 8)
			endingBit = startingBit + BATCH_SIZE * 8;
		else
			endingBit = fileSize * 8;

		unsigned char workingChunk[CHUNK_SIZE];
		unsigned char hash[20];

		unsigned int dataOffset = ((startingBit >> 3) / CHUNK_SIZE) * CHUNK_SIZE;
		unsigned int currentChunkSize;

		if (dataOffset < fileSize - CHUNK_SIZE)
			currentChunkSize = CHUNK_SIZE;
		else
			currentChunkSize = fileSize - dataOffset;

		memcpy(workingChunk, &pieceData[dataOffset], currentChunkSize);

		SHA1_CTX cachedCtx = midstates[(startingBit >> 3) / CHUNK_SIZE];

		for (unsigned int bitIdx = startingBit; bitIdx < endingBit; bitIdx++) {
			if (*result != -1) 
				return;

			SHA1_CTX ctx = cachedCtx;

			if (bitIdx != startingBit) {
				workingChunk[((bitIdx - 1) >> 3) % CHUNK_SIZE] ^= (1 << ((bitIdx - 1) % 8));
			}

			workingChunk[(bitIdx >> 3) % CHUNK_SIZE] ^= (1 << (bitIdx % 8));

			memcpy(ctx.data, workingChunk, CHUNK_SIZE);
			sha1_transform(&ctx, workingChunk);
			ctx.bitlen += 512;

			if (endingBit != fileSize * 8) {
				for (unsigned int chunkOffset = dataOffset + CHUNK_SIZE; chunkOffset < fileSize; chunkOffset += CHUNK_SIZE) {
					sha1_transform(&ctx, &pieceData[chunkOffset]);
					ctx.bitlen += 512;
				}
				memcpy(ctx.data, &pieceData[fileSize - CHUNK_SIZE], CHUNK_SIZE);
			}

			ctx.datalen = 0;

			sha1_final(&ctx, hash);

			if (cuda_bytecmp(hash, pieceHash)) {
				// printf("Result: %d\n", bitIdx);
				*result = bitIdx;
			}
		}
	}		
}

__device__ __forceinline__ int cuda_bytecmp(register const unsigned char* s1, register const unsigned char* s2) {
	register unsigned char n = 15;
	do {
		if (*s1 != *s2++)
			return 0;
		if (*s1++ == 0)
			break;
	} while (--n != 0);
	return 1;
}