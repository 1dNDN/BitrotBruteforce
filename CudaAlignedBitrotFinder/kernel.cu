#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <stdexcept>
#include <iostream>
#include <fstream>

#include "sha1.cuh"

extern "C" {
    void __declspec(dllexport) bruteforceBits(unsigned char* pieceData, unsigned char* pieceHash, size_t pieceSize, unsigned int* result)
    {
        unsigned char* dev_pieceData = 0;
        unsigned char* dev_pieceHash = 0;
        SHA1_CTX* dev_midstates = 0;
        unsigned int* dev_result = 0;
        hipError_t cudaStatus;

        size_t midstatesLength = pieceSize / CHUNK_SIZE;
        SHA1_CTX* midstates = new SHA1_CTX[midstatesLength + 1];

        SHA1_CTX ctx;
        sha1_init(&ctx);

        midstates[0] = ctx;
        for (int i = 0; i < midstatesLength; i++) {
            sha1_update(&ctx, &pieceData[i * CHUNK_SIZE], CHUNK_SIZE);
            midstates[i + 1] = ctx;
        }

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!");
            goto Error;
        }

        // alloc
        cudaStatus = hipMalloc((void**)&dev_pieceData, pieceSize);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_pieceHash, 20);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_midstates, ((pieceSize / CHUNK_SIZE) + 1) * sizeof(SHA1_CTX));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_result, sizeof(unsigned int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }


        // copy
        cudaStatus = hipMemcpy(dev_pieceData, pieceData, pieceSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_pieceHash, pieceHash, 20, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_midstates, midstates, ((pieceSize / CHUNK_SIZE) + 1) * sizeof(SHA1_CTX), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_result, result, sizeof(unsigned int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        int threadsPerBlock = 1024;
        int blocksPerGrid = ((pieceSize / BATCH_SIZE) + threadsPerBlock - 1) / threadsPerBlock;
        bitFlipKernel << <blocksPerGrid, threadsPerBlock >> > (dev_pieceData, dev_pieceHash, dev_midstates, pieceSize, dev_result);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
            goto Error;
        }

        cudaStatus = hipMemcpy(result, dev_result, sizeof(unsigned int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

    Error:
        hipFree(dev_pieceData);
        hipFree(dev_pieceHash);
        hipFree(dev_midstates);
        hipFree(dev_result);

        delete[] midstates;

        return;
    }
}

unsigned char* hexStringToBytes(const char* hexStr, size_t& byteArrayLength) {
    size_t hexStrLength = std::strlen(hexStr);

    if (hexStrLength % 2 != 0) {
        return nullptr;
    }

    byteArrayLength = hexStrLength / 2;

    unsigned char* byteArray = new unsigned char[byteArrayLength];

    for (size_t i = 0; i < byteArrayLength; ++i) {
        char byteString[3] = { hexStr[2 * i], hexStr[2 * i + 1], '\0' };
        byteArray[i] = static_cast<unsigned char>(std::strtoul(byteString, nullptr, 16));
    }

    return byteArray;
}

int main(int argc, char** argv)
{
    if (argc != 3) {
        std::cerr << "Error: Not enough arguments supplied! Usage: " << argv[0] << " <piece path> " << "<expected hash>" << std::endl;
        return 1;
    }

    size_t byteArrayLength = 0;
    auto pieceHash = hexStringToBytes(argv[2], byteArrayLength);

    if (byteArrayLength != 20) {
        std::cerr << "Error: Incorrect expected hash length";
        return 1;
    }

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <file_path>" << std::endl;
        return 1;
    }

    std::string piecePath = argv[1];

    std::ifstream file(piecePath, std::ios::binary | std::ios::ate);
    if (!file) {
        std::cerr << "Error: File '" << piecePath << "' does not exist or cannot be opened." << std::endl;
        return 1;
    }

    std::streamsize fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    unsigned char* fileData = new unsigned char[fileSize];

    if (!file.read(reinterpret_cast<char*>(fileData), fileSize)) {
        std::cerr << "Error: Failed to read the file." << std::endl;
        delete[] fileData;
        return 1;
    }

    file.close();

    std::cout << "File size: " << fileSize << " bytes" << std::endl;
    std::cout << "First few bytes: ";
    for (size_t i = 0; i < std::min(fileSize, static_cast<std::streamsize>(64)); ++i) {
        std::cout << std::hex << static_cast<int>(fileData[i]) << " ";
    }
    std::cout << std::dec << std::endl;
   
    unsigned int result = -1;
    bruteforceBits(fileData, pieceHash, fileSize, &result);

    std::cout << "Result: " << result << std::endl;

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    delete[] pieceHash;
    delete[] fileData;

    return 0;
}

