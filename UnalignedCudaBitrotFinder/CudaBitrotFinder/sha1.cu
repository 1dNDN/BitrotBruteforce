#include "hip/hip_runtime.h"
#include "sha1.cuh"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif

constexpr auto K0 = 0x5a827999;
constexpr auto K1 = 0x6ed9eba1;
constexpr auto K2 = 0x8f1bbcdc;
constexpr auto K3 = 0xca62c1d6;

__host__ __device__ __forceinline__ void sha1_transform(SHA1_CTX* ctx, const BYTE data[])
{
	WORD a, b, c, d, e, i, j, t, m[80];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) + (data[j + 1] << 16) + (data[j + 2] << 8) + (data[j + 3]);
	for (; i < 80; ++i) {
		m[i] = (m[i - 3] ^ m[i - 8] ^ m[i - 14] ^ m[i - 16]);
		m[i] = (m[i] << 1) | (m[i] >> 31);
	}

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];

	for (i = 0; i < 20; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (~b & d)) + e + K0 + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 40; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + K1 + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 60; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (b & d) ^ (c & d)) + e + K2 + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 80; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + K3 + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
}

__host__ __device__ void sha1_init(SHA1_CTX* ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xc3d2e1f0;
}

__host__ __device__ void sha1_update(SHA1_CTX* ctx, const BYTE data[], size_t len)
{
	size_t i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			sha1_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

 __host__ __device__ void sha1_final(SHA1_CTX* ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		sha1_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	sha1_transform(ctx, ctx->data);

	for (i = 0; i < 4; ++i) {
		hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
	}
}

__global__ void bitFlipKernel(unsigned char* pieceData, unsigned char* pieceHash, SHA1_CTX* midstates, size_t fileSize, unsigned int* result)
{
	 unsigned int bitIdx = blockIdx.x * blockDim.x + threadIdx.x;
	 if (bitIdx < fileSize * 8 && *result == -1) {
		 unsigned int byteIdx = bitIdx >> 3;
		 unsigned char workingChunk[CHUNK_SIZE];
		 unsigned char hash[20];
		 unsigned int dataOffset = (byteIdx / CHUNK_SIZE) * CHUNK_SIZE;
		 unsigned int currentChunkSize;

		 if (dataOffset < fileSize - CHUNK_SIZE)
			currentChunkSize = CHUNK_SIZE;
		 else
			currentChunkSize = fileSize - dataOffset;

		 memcpy(workingChunk, &pieceData[dataOffset], currentChunkSize);

		 SHA1_CTX ctx = midstates[byteIdx / CHUNK_SIZE];

		 workingChunk[byteIdx % CHUNK_SIZE] ^= (1 << (bitIdx % 8));

		 sha1_update(&ctx, workingChunk, currentChunkSize);

		 if (fileSize - dataOffset > CHUNK_SIZE) {
			 sha1_update(&ctx, &pieceData[dataOffset + CHUNK_SIZE], fileSize - (dataOffset + CHUNK_SIZE));
		 }

		 sha1_final(&ctx, hash);

		 if (cuda_bytecmp(hash, pieceHash)) {
			 // printf("Found result: %d\n", bitIdx);
			 *result = bitIdx;
		 }
	 }
 }

__device__ __forceinline__ int cuda_bytecmp(register const unsigned char* s1, register const unsigned char* s2) {
	register unsigned char n = 15;
	do {
		if (*s1 != *s2++)
			return 0;
		if (*s1++ == 0)
			break;
	} while (--n != 0);
	return 1;
}